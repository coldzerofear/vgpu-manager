
#include <hip/hip_runtime.h>
#include <iostream>
#include <csignal>
#include <cstdlib>
#include <cmath> // 引入数学库，用于增加计算复杂度

// 定义一个简单的信号处理函数，用于捕获Ctrl+C信号并退出程序
void signalHandler(int signum) {
    std::cout << "捕获到信号 " << signum << "，程序将退出..." << std::endl;
    hipDeviceReset(); // 重置设备，释放所有资源
    exit(signum); // 退出程序
}

// 定义矩阵的大小
const int N = 2048; // 增大矩阵大小，增加计算量
const int NUM_ITERATIONS = 99999999; // 迭代次数，可以根据需要调整

// CUDA内核函数，用于矩阵加法和一些额外的复杂计算
__global__ void matrixAddAndCompute(float* A, float* B, float* C, int iterations) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i * N + j;

    for (int iter = 0; iter < iterations; iter++) {
       // if (i < N && j < N) {
            C[index] += A[index] + B[index]; // 多次累加，增加计算量

            // 添加一些额外的复杂计算，比如计算平方根和三角函数
            float temp = sqrt(C[index]) * sin(C[index]);
            // 这里我们并不使用temp的结果，只是为了增加计算复杂度
        //}
    }
}

int main() {
    // 注册信号处理函数
    signal(SIGINT, signalHandler);

    // 分配和初始化主机内存
    float* h_A = (float*)malloc(N * N * sizeof(float));
    float* h_B = (float*)malloc(N * N * sizeof(float));
    for (int i = 0; i < N * N; i++) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // 分配设备内存
    float* d_A = nullptr;
    float* d_B = nullptr;
    float* d_C = nullptr;
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));

    // 初始化设备上的C矩阵为0
    hipMemset(d_C, 0, N * N * sizeof(float));

    // 将数据从主机复制到设备
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // 定义块和网格的大小
    dim3 blockSize(1024, 1024);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    // 无限循环，直到捕获到Ctrl+C信号
    while (true) {
        // 执行矩阵加法运算和额外的复杂计算
        matrixAddAndCompute<<<gridSize, blockSize>>>(d_A, d_B, d_C, NUM_ITERATIONS);

        // 同步设备，确保计算完成
        hipDeviceSynchronize();

        // 你可以在这里添加一些其他的计算或操作，但请注意，这仍然是一个无限循环
    }

    // 注意：由于存在无限循环，下面的代码实际上不会被执行，除非在循环中添加适当的退出条件
    // 释放设备内存
    // cudaFree(d_A);
    // cudaFree(d_B);
    // cudaFree(d_C);

    // 释放主机内存
    // free(h_A);
    // free(h_B);

    return 0;
}