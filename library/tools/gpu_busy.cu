
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void busyKernel() {
    while (true) {
        volatile int x = 1;
        x = x * 2;
    }
}

int main() {
    printf("Starting GPU busy kernel...\n");

    int numBlocks = 1;
    int threadsPerBlock = 1;

    busyKernel<<<numBlocks, threadsPerBlock>>>();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    printf("GPU is now busy. Press Ctrl+C to terminate the program.\n");

    while (true) {}

    return 0;
}