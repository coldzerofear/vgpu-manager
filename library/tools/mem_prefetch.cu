#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define N 1000000

// 初始化数据的核函数
__global__ void initKernel(int* data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data[idx] = idx;
    }
}

int main() {
    int *data;
    hipStream_t stream;

    // 1. 分配统一内存
    hipMallocManaged(&data, N * sizeof(int));

    printf("hipMallocManaged\n");
    sleep(10);

    // 2. 创建CUDA流
    hipStreamCreate(&stream);

    // 3. 预取数据到当前GPU (异步)
    hipMemPrefetchAsync(data, N * sizeof(int), 0, stream); // 0 表示当前GPU

    // 4. 在流中执行初始化内核
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    initKernel<<<grid, block, 0, stream>>>(data);

    printf("hipMemPrefetchAsync to gpu\n");
    sleep(10);

    // 5. 预取回CPU以便后续处理
    hipMemPrefetchAsync(data, N * sizeof(int), hipCpuDeviceId, stream);

    // 6. 同步流确保操作完成
    hipStreamSynchronize(stream);

    printf("hipMemPrefetchAsync to cpu\n");
    sleep(10);

    // 7. 验证结果 (CPU访问)
    for (int i = 0; i < 10; i++) {
        printf("data[%d] = %d\n", i, data[i]);
    }

    // 8. 清理资源
    hipFree(data);
    hipStreamDestroy(stream);

    return 0;
}